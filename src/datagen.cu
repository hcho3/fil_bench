#include "hip/hip_runtime.h"
#include <fil_bench/datagen.hpp>
#include <fil_bench/array_types.hpp>

#include <raft/core/handle.hpp>
#include <raft/random/make_regression.cuh>
#include <treelite/tree.h>
#include <cuml/ensemble/randomforest.hpp>

#include <cstdint>
#include <memory>
#include <utility>

namespace fil_bench {

std::pair<Device2DArray, Device1DArray> make_regression(const raft::handle_t& handle,
    std::uint64_t n_rows, std::uint64_t n_cols) {
  auto [X, y] = make_empty(handle, n_rows, n_cols);

  raft::random::make_regression(handle,
      X.data_handle(),
      y.data_handle(),
      n_rows,
      n_cols,
      n_cols / 3,
      handle.get_stream(),
      (float*)nullptr,
      std::uint64_t(1),
      0.0f,
      n_cols / 3,
      0.1f,
      0.01f,
      false,
      12345ULL);
  handle.sync_stream();
  handle.sync_stream_pool();

  return {X, y};
}

std::pair<Device2DArray, Device1DArray> make_empty(
    raft::handle_t const& handle, std::uint64_t n_rows, std::uint64_t n_cols) {
  Device2DArray X = raft::make_device_matrix<float>(handle, n_rows, n_cols);
  Device1DArray y = raft::make_device_vector<float>(handle, n_rows);
  return {X, y};
}

std::unique_ptr<treelite::Model> fit_rf_regressor(
    const raft::handle_t& handle, Device2DArrayView X, Device1DArrayView y,
    std::uint32_t n_trees, std::uint32_t max_depth) {
  // Take first 1000 rows as training set
  auto train_nrows = std::min(X.extent(0), std::uint64_t(1000));
  auto rf_model = std::make_unique<ML::RandomForestRegressorF>();
  auto* rf_model_ptr = rf_model.get();
  ML::RF_params rf_params = ML::set_rf_params(
      static_cast<int>(max_depth),
      (1 << 20),
      1.f,
      32,
      3,
      3,
      0.0f,
      true,
      static_cast<int>(n_trees),
      1.f,
      1234ULL,
      ML::CRITERION::MSE,
      8,
      128
  );
  ML::fit(handle, rf_model_ptr, X.data_handle(), train_nrows, X.extent(1), y.data_handle(), rf_params);
  handle.sync_stream();
  handle.sync_stream_pool();

  void* tl_model_ptr{nullptr};
  ML::build_treelite_forest(&tl_model_ptr, rf_model.get(), X.extent(1));

  return std::unique_ptr<treelite::Model>{static_cast<treelite::Model*>(tl_model_ptr)};
}

}  // namespace fil_bench
