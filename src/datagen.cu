#include "hip/hip_runtime.h"
#include <fil_bench/datagen.hpp>
#include <fil_bench/array_types.hpp>
#include <fil_bench/constants.hpp>

#include <raft/core/handle.hpp>
#include <raft/random/make_regression.cuh>
#include <treelite/tree.h>
#include <cuml/ensemble/randomforest.hpp>

#include <memory>
#include <utility>

namespace fil_bench {

std::pair<Device2DArray, Device1DArray> make_regression(const raft::handle_t& handle) {
  Device2DArray X = raft::make_device_matrix<float>(handle, nrows, ncols);
  auto y = raft::make_device_vector<float>(handle, nrows);

  raft::random::make_regression(handle,
      X.data_handle(),
      y.data_handle(),
      nrows,
      ncols,
      6,
      handle.get_stream(),
      (float*)nullptr,
      1,
      0.0f,
      6,
      0.1f,
      0.01f,
      false,
      12345ULL);
  handle.sync_stream();
  handle.sync_stream_pool();

  return {X, y};
}

std::unique_ptr<treelite::Model> fit_rf_regressor(
    const raft::handle_t& handle, Device2DArrayView X, Device1DArrayView y) {
  // Take first 1000 rows as training set
  auto train_nrows = std::min(nrows, 1000);
  auto rf_model = std::make_unique<ML::RandomForestRegressorF>();
  auto* rf_model_ptr = rf_model.get();
  ML::RF_params rf_params = ML::set_rf_params(10,
      (1 << 20),
      1.f,
      32,
      3,
      3,
      0.0f,
      true,
      1,
      1.f,
      1234ULL,
      ML::CRITERION::MSE,
      8,
      128
  );
  ML::fit(handle, rf_model_ptr, X.data_handle(), train_nrows, ncols, y.data_handle(), rf_params);
  handle.sync_stream();
  handle.sync_stream_pool();

  void* tl_model_ptr{nullptr};
  ML::build_treelite_forest(&tl_model_ptr, rf_model.get(), ncols);

  return std::unique_ptr<treelite::Model>{static_cast<treelite::Model*>(tl_model_ptr)};
}

}  // namespace fil_bench
